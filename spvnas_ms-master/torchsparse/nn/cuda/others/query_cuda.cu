#include "hip/hip_runtime.h"
#include <torch/torch.h>

#include <cmath>
#include <iostream>
#include <vector>
#include "ms_ext.h"

#include "../hashmap/hashmap_cuda.cuh"

void hash_query_cuda(const at::Tensor hash_query,
                           const at::Tensor hash_target,
                           const at::Tensor idx_target,
                           at::Tensor out) {
  // return group_point_forward_gpu(points, indices);
  int n = hash_target.size(0);
  int n1 = hash_query.size(0);
  const int nextPow2 = pow(2, ceil(log2((double)n)));
  // When n is large, the hash values tend to be more evenly distrubuted and
  // choosing table_size to be 2 * nextPow2 typically suffices. For smaller n,
  // the effect of uneven distribution of hash values is more pronounced and
  // hence we choose table_size to be 4 * nextPow2 to reduce the chance of
  // bucket overflow.
  int table_size = (n < 2048) ? 4 * nextPow2 : 2 * nextPow2;
  if (table_size < 512) {
    table_size = 512;
  }
  int num_funcs = 3;
  CuckooHashTableCuda_Multi in_hash_table(table_size, 8 * ceil(log2((double)n)),
                                          num_funcs);
  at::Tensor key_buf =
      torch::zeros({table_size},
                   at::device(hash_query.device()).dtype(at::ScalarType::Long));
  at::Tensor val_buf =
      torch::zeros({table_size},
                   at::device(hash_query.device()).dtype(at::ScalarType::Long));
  at::Tensor key =
      torch::zeros({num_funcs * table_size},
                   at::device(hash_query.device()).dtype(at::ScalarType::Long));
  at::Tensor val =
      torch::zeros({num_funcs * table_size},
                   at::device(hash_query.device()).dtype(at::ScalarType::Long));

  in_hash_table.insert_vals((uint64_t *)(hash_target.data_ptr<int64_t>()),
                            (uint64_t *)(idx_target.data_ptr<int64_t>()),
                            (uint64_t *)(key_buf.data_ptr<int64_t>()),
                            (uint64_t *)(val_buf.data_ptr<int64_t>()),
                            (uint64_t *)(key.data_ptr<int64_t>()),
                            (uint64_t *)(val.data_ptr<int64_t>()), n);

  // at::Tensor out = torch::zeros(
  //     {n1}, at::device(hash_query.device()).dtype(at::ScalarType::Long));

  in_hash_table.lookup_vals((uint64_t *)(hash_query.data_ptr<int64_t>()),
                            (uint64_t *)(key.data_ptr<int64_t>()),
                            (uint64_t *)(val.data_ptr<int64_t>()),
                            (uint64_t *)(out.data_ptr<int64_t>()), n1);
  // return out;
}

extern "C" int hash_query_ms(int nparam, void** params, int* ndims, int64_t** shapes, 
                        const char** dtypes, void* stream, void* extra) {
  hipStream_t custream = static_cast<hipStream_t>(stream);
  hipStreamSynchronize(custream);

  // transform ms tensor to pytorch tensor
  auto tensors = get_torch_tensors(nparam, params, ndims, shapes, dtypes, c10::kCUDA);
  auto hash_query = tensors[0];
  auto hash_target = tensors[1];
  auto idx_target = tensors[2];
  auto output = tensors[3];
  
  // Do the computation
  hash_query_cuda(hash_query, hash_target, idx_target, output);

  return 0;
}
